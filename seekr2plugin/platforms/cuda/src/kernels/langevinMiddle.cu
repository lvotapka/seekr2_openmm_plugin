#include "hip/hip_runtime.h"
enum {VelScale, NoiseScale};

/**
 * Perform the first step of Langevin Middle integration: velocity step.
 */

extern "C" __global__ void integrateMmvtLangevinMiddlePart1(int numAtoms, 
            int paddedNumAtoms, mixed4* __restrict__ velm, 
            const long long* __restrict__ force, 
            const mixed2* __restrict__ dt) {
    mixed fscale = dt[0].y/(mixed) 0x100000000;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numAtoms; index += blockDim.x*gridDim.x) {
        mixed4 velocity = velm[index];
        if (velocity.w != 0.0) {
            velocity.x += fscale*velocity.w*force[index];
            velocity.y += fscale*velocity.w*force[index+paddedNumAtoms];
            velocity.z += fscale*velocity.w*force[index+paddedNumAtoms*2];
            velm[index] = velocity;
        }
    }
}

/**
 * Perform the second part of integration: position half step, then interact with heat bath,
 * then another position half step.
 */

extern "C" __global__ void integrateMmvtLangevinMiddlePart2(int numAtoms, 
        mixed4* __restrict__ velm, mixed4* __restrict__ posDelta,
        mixed4* __restrict__ oldDelta, const mixed* __restrict__ paramBuffer, 
        const mixed2* __restrict__ dt, mixed4* __restrict__ oldVelm,
        const float4* __restrict__ random, unsigned int randomIndex
        ) {
    mixed vscale = paramBuffer[VelScale];
    mixed noisescale = paramBuffer[NoiseScale];
    mixed halfdt = 0.5f*dt[0].y;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    randomIndex += index;
    while (index < numAtoms) {
        mixed4 velocity = velm[index];
        oldVelm[index] = velm[index];
        
        if (velocity.w != 0.0) {
            mixed4 delta = make_mixed4(halfdt*velocity.x, halfdt*velocity.y, halfdt*velocity.z, 0);
            mixed sqrtInvMass = SQRT(velocity.w);
            velocity.x = vscale*velocity.x + noisescale*sqrtInvMass*random[randomIndex].x;
            velocity.y = vscale*velocity.y + noisescale*sqrtInvMass*random[randomIndex].y;
            velocity.z = vscale*velocity.z + noisescale*sqrtInvMass*random[randomIndex].z;
            velm[index] = velocity;
            //delta = delta + make_mixed4(halfdt*velocity.x, halfdt*velocity.y, halfdt*velocity.z, 0);
            delta.x += (mixed) halfdt*velocity.x;
            delta.y += (mixed) halfdt*velocity.y;
            delta.z += (mixed) halfdt*velocity.z;
            posDelta[index] = delta;
            oldDelta[index] = delta;
        }
        
        randomIndex += blockDim.x*gridDim.x;
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Perform the third part of integration: apply constraint forces to velocities, then record
 * the constrained positions.
 */

extern "C" __global__ void integrateMmvtLangevinMiddlePart3(int numAtoms, 
         real4* __restrict__ posq, mixed4* __restrict__ velm,
         const mixed4* __restrict__ posDelta, mixed4* __restrict__ oldDelta, 
         const mixed2* __restrict__ dt, real4* __restrict__ oldPosq, 
         real4* __restrict__ posqCorrection) {
    mixed invDt = 1/dt[0].y;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numAtoms; index += blockDim.x*gridDim.x) {
        mixed4 velocity = velm[index];
        oldPosq[index] = posq[index];
        if (velocity.w != 0.0) {
            mixed4 delta = posDelta[index];
            velocity.x += (delta.x-oldDelta[index].x)*invDt;
            velocity.y += (delta.y-oldDelta[index].y)*invDt;
            velocity.z += (delta.z-oldDelta[index].z)*invDt;
            velm[index] = velocity;
#ifdef USE_MIXED_PRECISION
            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
        }
    }
}


extern "C" __global__ void integrateElberLangevinMiddlePart1(int numAtoms, 
            int paddedNumAtoms, mixed4* __restrict__ velm, 
            const long long* __restrict__ force, 
            const mixed2* __restrict__ dt) {
    mixed fscale = dt[0].y/(mixed) 0x100000000;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numAtoms; index += blockDim.x*gridDim.x) {
        mixed4 velocity = velm[index];
        if (velocity.w != 0.0) {
            velocity.x += fscale*velocity.w*force[index];
            velocity.y += fscale*velocity.w*force[index+paddedNumAtoms];
            velocity.z += fscale*velocity.w*force[index+paddedNumAtoms*2];
            velm[index] = velocity;
        }
    }
}

/**
 * Perform the second part of integration: position half step, then interact with heat bath,
 * then another position half step.
 */

extern "C" __global__ void integrateElberLangevinMiddlePart2(int numAtoms, 
        mixed4* __restrict__ velm, mixed4* __restrict__ posDelta,
        mixed4* __restrict__ oldDelta, const mixed* __restrict__ paramBuffer, 
        const mixed2* __restrict__ dt,
        const float4* __restrict__ random, unsigned int randomIndex
        ) {
    mixed vscale = paramBuffer[VelScale];
    mixed noisescale = paramBuffer[NoiseScale];
    mixed halfdt = 0.5f*dt[0].y;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    randomIndex += index;
    while (index < numAtoms) {
        mixed4 velocity = velm[index];
        
        if (velocity.w != 0.0) {
            mixed4 delta = make_mixed4(halfdt*velocity.x, halfdt*velocity.y, halfdt*velocity.z, 0);
            mixed sqrtInvMass = SQRT(velocity.w);
            velocity.x = vscale*velocity.x + noisescale*sqrtInvMass*random[randomIndex].x;
            velocity.y = vscale*velocity.y + noisescale*sqrtInvMass*random[randomIndex].y;
            velocity.z = vscale*velocity.z + noisescale*sqrtInvMass*random[randomIndex].z;
            velm[index] = velocity;
            //delta = delta + make_mixed4(halfdt*velocity.x, halfdt*velocity.y, halfdt*velocity.z, 0);
            delta.x += (mixed) halfdt*velocity.x;
            delta.y += (mixed) halfdt*velocity.y;
            delta.z += (mixed) halfdt*velocity.z;
            posDelta[index] = delta;
            oldDelta[index] = delta;
        }
        
        randomIndex += blockDim.x*gridDim.x;
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Perform the third part of integration: apply constraint forces to velocities, then record
 * the constrained positions.
 */

extern "C" __global__ void integrateElberLangevinMiddlePart3(int numAtoms, 
         real4* __restrict__ posq, mixed4* __restrict__ velm,
         const mixed4* __restrict__ posDelta, mixed4* __restrict__ oldDelta, 
         const mixed2* __restrict__ dt,
         real4* __restrict__ posqCorrection) {
    mixed invDt = 1/dt[0].y;
    for (int index = blockIdx.x*blockDim.x+threadIdx.x; index < numAtoms; index += blockDim.x*gridDim.x) {
        mixed4 velocity = velm[index];
        if (velocity.w != 0.0) {
            mixed4 delta = posDelta[index];
            velocity.x += (delta.x-oldDelta[index].x)*invDt;
            velocity.y += (delta.y-oldDelta[index].y)*invDt;
            velocity.z += (delta.z-oldDelta[index].z)*invDt;
            velm[index] = velocity;
#ifdef USE_MIXED_PRECISION
            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
        }
    }
}


/**
 * Take a step back in time and reverse velocities
 */
extern "C" __global__ void mmvtBounce(int numAtoms, int paddedNumAtoms, 
            real4* __restrict__ posq,
            mixed4* __restrict__ velm, 
            const real4* __restrict__ oldPosq, 
            const mixed4* __restrict__ oldVelm) { 
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    while (index < numAtoms) {
        posq[index] = oldPosq[index];
        velm[index] = make_mixed4(-oldVelm[index].x, -oldVelm[index].y,
                -oldVelm[index].z, oldVelm[index].w);
        index += blockDim.x*gridDim.x;
    }
}