#include "hip/hip_runtime.h"
enum {VelScale, ForceScale, NoiseScale, MaxParams};

/**
 * Perform the first step of Langevin integration.
 */

extern "C" __global__ void integrateMmvtLangevinPart1(int numAtoms, 
            int paddedNumAtoms, mixed4* __restrict__ velm, 
            const long long* __restrict__ force, mixed4* __restrict__ posDelta,
            const mixed* __restrict__ paramBuffer, 
            const mixed2* __restrict__ dt, const float4* __restrict__ random, 
            unsigned int randomIndex) {
    mixed vscale = paramBuffer[VelScale];
    mixed fscale = paramBuffer[ForceScale]/(mixed) 0x100000000;
    mixed noisescale = paramBuffer[NoiseScale];
    mixed stepSize = dt[0].y;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    randomIndex += index;
    while (index < numAtoms) {
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            mixed sqrtInvMass = SQRT(velocity.w);
            velocity.x = vscale*velocity.x + fscale*velocity.w*force[index] + noisescale*sqrtInvMass*random[randomIndex].x;
            velocity.y = vscale*velocity.y + fscale*velocity.w*force[index+paddedNumAtoms] + noisescale*sqrtInvMass*random[randomIndex].y;
            velocity.z = vscale*velocity.z + fscale*velocity.w*force[index+paddedNumAtoms*2] + noisescale*sqrtInvMass*random[randomIndex].z;
            velm[index] = velocity;
            posDelta[index] = make_mixed4(stepSize*velocity.x, stepSize*velocity.y, stepSize*velocity.z, 0);
        }
        randomIndex += blockDim.x*gridDim.x;
        index += blockDim.x*gridDim.x;
    }
}

extern "C" __global__ void integrateElberLangevinPart1(int numAtoms, 
            int paddedNumAtoms, mixed4* __restrict__ velm, 
            const long long* __restrict__ force, mixed4* __restrict__ posDelta,
            const mixed* __restrict__ paramBuffer, 
            const mixed2* __restrict__ dt, const float4* __restrict__ random, 
            unsigned int randomIndex) {
    mixed vscale = paramBuffer[VelScale];
    mixed fscale = paramBuffer[ForceScale]/(mixed) 0x100000000;
    mixed noisescale = paramBuffer[NoiseScale];
    mixed stepSize = dt[0].y;
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    randomIndex += index;
    while (index < numAtoms) {
        mixed4 velocity = velm[index];
        if (velocity.w != 0) {
            mixed sqrtInvMass = SQRT(velocity.w);
            velocity.x = vscale*velocity.x + fscale*velocity.w*force[index] + noisescale*sqrtInvMass*random[randomIndex].x;
            velocity.y = vscale*velocity.y + fscale*velocity.w*force[index+paddedNumAtoms] + noisescale*sqrtInvMass*random[randomIndex].y;
            velocity.z = vscale*velocity.z + fscale*velocity.w*force[index+paddedNumAtoms*2] + noisescale*sqrtInvMass*random[randomIndex].z;
            velm[index] = velocity;
            posDelta[index] = make_mixed4(stepSize*velocity.x, stepSize*velocity.y, stepSize*velocity.z, 0);
        }
        randomIndex += blockDim.x*gridDim.x;
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Perform the second step of Langevin integration.
 */

extern "C" __global__ void integrateMmvtLangevinPart2(int numAtoms, 
            real4* __restrict__ posq, real4* __restrict__ posqCorrection, 
            const mixed4* __restrict__ posDelta, mixed4* __restrict__ velm, 
            const mixed2* __restrict__ dt, real4* __restrict__ oldPosq,
            mixed4* __restrict__ oldVelm) {
#if __CUDA_ARCH__ >= 130
    double invStepSize = 1.0/dt[0].y;
#else
    float invStepSize = 1.0f/dt[0].y;
    float correction = (1.0f-invStepSize*dt[0].y)/dt[0].y;
#endif
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    while (index < numAtoms) {
        mixed4 vel = velm[index];
        oldPosq[index] = posq[index];
        oldVelm[index] = velm[index];
        if (vel.w != 0) {
#ifdef USE_MIXED_PRECISION
            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            mixed4 delta = posDelta[index];
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
#if __CUDA_ARCH__ >= 130
            vel.x = (mixed) (invStepSize*delta.x);
            vel.y = (mixed) (invStepSize*delta.y);
            vel.z = (mixed) (invStepSize*delta.z);
#else
            vel.x = invStepSize*delta.x + correction*delta.x;
            vel.y = invStepSize*delta.y + correction*delta.x;
            vel.z = invStepSize*delta.z + correction*delta.x;
#endif
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
            velm[index] = vel;
        }
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Perform the second step of Langevin integration.
 */

extern "C" __global__ void integrateElberLangevinPart2(int numAtoms, 
            real4* __restrict__ posq, real4* __restrict__ posqCorrection, 
            const mixed4* __restrict__ posDelta, mixed4* __restrict__ velm, 
            const mixed2* __restrict__ dt) {
#if __CUDA_ARCH__ >= 130
    double invStepSize = 1.0/dt[0].y;
#else
    float invStepSize = 1.0f/dt[0].y;
    float correction = (1.0f-invStepSize*dt[0].y)/dt[0].y;
#endif
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    while (index < numAtoms) {
        mixed4 vel = velm[index];
        if (vel.w != 0) {
#ifdef USE_MIXED_PRECISION
            real4 pos1 = posq[index];
            real4 pos2 = posqCorrection[index];
            mixed4 pos = make_mixed4(pos1.x+(mixed)pos2.x, pos1.y+(mixed)pos2.y, pos1.z+(mixed)pos2.z, pos1.w);
#else
            real4 pos = posq[index];
#endif
            mixed4 delta = posDelta[index];
            pos.x += delta.x;
            pos.y += delta.y;
            pos.z += delta.z;
#if __CUDA_ARCH__ >= 130
            vel.x = (mixed) (invStepSize*delta.x);
            vel.y = (mixed) (invStepSize*delta.y);
            vel.z = (mixed) (invStepSize*delta.z);
#else
            vel.x = invStepSize*delta.x + correction*delta.x;
            vel.y = invStepSize*delta.y + correction*delta.x;
            vel.z = invStepSize*delta.z + correction*delta.x;
#endif
#ifdef USE_MIXED_PRECISION
            posq[index] = make_real4((real) pos.x, (real) pos.y, (real) pos.z, (real) pos.w);
            posqCorrection[index] = make_real4(pos.x-(real) pos.x, pos.y-(real) pos.y, pos.z-(real) pos.z, 0);
#else
            posq[index] = pos;
#endif
            velm[index] = vel;
        }
        index += blockDim.x*gridDim.x;
    }
}

/**
 * Take a step back in time and reverse velocities
 */
extern "C" __global__ void mmvtBounce(int numAtoms, int paddedNumAtoms, 
            real4* __restrict__ posq,
            mixed4* __restrict__ velm, 
            const real4* __restrict__ oldPosq, 
            const mixed4* __restrict__ oldVelm) { 
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    while (index < numAtoms) {
        posq[index] = oldPosq[index];
        velm[index] = make_mixed4(-oldVelm[index].x, -oldVelm[index].y,
                -oldVelm[index].z, oldVelm[index].w);
        index += blockDim.x*gridDim.x;
    }
}